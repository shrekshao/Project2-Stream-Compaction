#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
	namespace Efficient {
		//const int blockSize = 128;

		

		__global__ void kernUpSweep(int size, int step, int * data)
		{
			//step = 2^(d+1)
			int k = threadIdx.x + blockDim.x * blockIdx.x;
			
			if(k < size)
			{
				if ( k % step == 0 )
				{
					data[k + step - 1] += data[k + (step>>1) - 1];
				}
			}
			
		}

		__global__ void kernDownSweep(int size,int step, int * data)
		{
			//step = 2^(d+1)
			int k = threadIdx.x + blockDim.x * blockIdx.x;

			if(k < size)
			{
				if ( k % step == 0 )
				{
					int left_child = data[k + (step>>1) - 1];
					data[k + (step>>1) - 1] = data[k + step - 1];
					data[k + step - 1] += left_child;
				}
			}
		}


		__global__ void kernSetRootZero(int rootId, int * data)
		{
			int k = threadIdx.x + blockDim.x * blockIdx.x;
			if(k == rootId)
			{
				data[k] = 0;
			}
		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata,bool is_dev_data) {
			//if using device data directly
			
			
			int * dev_data;

			int ceil_log2n = ilog2ceil(n);
			int size = 1 << ceil_log2n;

			dim3 fullBlocksPerGrid((size + blockSize - 1) / blockSize); 


			hipMalloc((void**)&dev_data, size * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed");
			Common::kernZeroArray<<< fullBlocksPerGrid, blockSize>>>(size, dev_data);
			if(!is_dev_data)
			{
				//host data
				hipMemcpy(dev_data,idata, n * sizeof(int),hipMemcpyHostToDevice);
				checkCUDAError("hipMemcpy from data to dev_data failed");
			}
			else
			{
				hipMemcpy(dev_data,idata, n * sizeof(int),hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy from data to dev_data failed");
			}
			hipDeviceSynchronize();

			//UpSweep
			for(int d = 0 ; d < ceil_log2n - 1 ; d++)
			{
				kernUpSweep<<< fullBlocksPerGrid, blockSize>>> (size, 1<<(d+1) , dev_data);
				hipDeviceSynchronize();
			}

			kernSetRootZero<<< fullBlocksPerGrid, blockSize>>> ( size - 1 , dev_data);
			hipDeviceSynchronize();
			
			for(int d = ceil_log2n - 1 ; d >= 0 ; d--)
			{
				kernDownSweep<<< fullBlocksPerGrid, blockSize>>> (size, 1<<(d+1) , dev_data);
				hipDeviceSynchronize();
			}


			if(!is_dev_data)
			{
				hipMemcpy(odata,dev_data,n * sizeof(int),hipMemcpyDeviceToHost);
				checkCUDAError("hipMemcpy from dev_data to odata failed");
			}
			else
			{
				hipMemcpy(odata,dev_data,n * sizeof(int),hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy from dev_data to odata failed");
			}
			hipFree(dev_data);
		}

		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {
			int hos_scans;
			int hos_bools;
			int * dev_bools;
			int * dev_scans;
			int * dev_idata;
			int * dev_odata;
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize); 

			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAError("hipMalloc dev_bools failed");
			hipMalloc((void**)&dev_scans, n * sizeof(int));
			checkCUDAError("hipMalloc dev_scans failed");
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed");

			hipMemcpy(dev_idata,idata, n * sizeof(int),hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from data to dev_data failed");
			hipDeviceSynchronize();

			Common::kernMapToBoolean<<< fullBlocksPerGrid, blockSize>>> ( n , dev_bools, dev_idata );
			hipDeviceSynchronize();

			//hipMemcpy(hos_bools,dev_bools, n * sizeof(int),hipMemcpyDeviceToHost);
			//checkCUDAError("hipMemcpy from data to dev_data failed");
			//hipDeviceSynchronize();

			scan(n,dev_scans,dev_bools,true);

			//hipMemcpy(dev_scans,hos_scans, n * sizeof(int),hipMemcpyHostToDevice);
			//checkCUDAError("hipMemcpy from hos_scans to dev_scans failed");
			//hipDeviceSynchronize();

			Common::kernScatter<<< fullBlocksPerGrid, blockSize>>>(n, dev_odata,
				dev_idata, dev_bools, dev_scans);
			hipDeviceSynchronize();

			hipMemcpy(odata,dev_odata,n * sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed");
			//hipDeviceSynchronize();

			hipMemcpy(&hos_scans,dev_scans+n-1,sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy scans[n-1] failed");

			hipMemcpy(&hos_bools,dev_bools+n-1,sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy bools[n-1] failed");

			hipDeviceSynchronize();



			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_bools);
			hipFree(dev_scans);

			//int num = hos_scans[n-1] + hos_bools[n-1];
			int num = hos_scans + hos_bools;
			//delete[] hos_scans;
			//delete[] hos_bools;

			return num;
		}







		//Radix sort


		__global__ void kernGetE(int n, int * odata, const int * idata,int cur_bit)
		{
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if( index < n)
			{
				odata[index] = 1 - ( ( idata[index] & (1 << cur_bit ) ) >> cur_bit   );
			}
		}

		__global__ void kernGetK(int n, int* t, const int * f, const int totalFalses)
		{
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if( index < n)
			{
				t[index] = index - f[index] + totalFalses;
			}
		}

		__global__ void kernRadixScatter(int n, int * odata,const int * idata, const int * e, const int * t, const int * f)
		{
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if( index < n)
			{
				 odata[  (e[index]==0) ? t[index] : f[index]  ] = idata[index] ;
			}
		}



		int * dev_i;
		int * dev_o;
		int * dev_e;	// dev_e[i] = 1 - dev_idata[i].cur_bit
		int * dev_f;	// exclusive scan of dev_e, id if false
		int * dev_t;	// i �Cf[i] + totalFalses, id if true
		

		

		void radixSort(int n, int *dev_odata, const int *dev_idata, int cur_bit)
		{
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			// get e
			kernGetE<<< fullBlocksPerGrid, blockSize>>>(n,dev_e,dev_idata,cur_bit);
			hipDeviceSynchronize();

			scan(n,dev_f,dev_e,true);
			int totalFalses;
			int last_e;
			hipMemcpy(&last_e,dev_e+n-1,sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_e[n-1] failed");
			hipMemcpy(&totalFalses,dev_f+n-1,sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_f[n-1] failed");
			totalFalses += last_e;

			//get t
			kernGetK<<< fullBlocksPerGrid, blockSize>>>(n,dev_t,dev_f,totalFalses);

			//scatter
			kernRadixScatter<<< fullBlocksPerGrid, blockSize>>>(n,dev_odata,dev_idata,dev_e,dev_t,dev_f);
		}



		//wrapper
		void radixSortLauncher(int n, int *odata, const int *idata, int msb,int lsb)
		{
			//simple version
			//no split, no merge, no shared memory
			
			//split
			 

			//sort
			//for each split
			
			
			hipMalloc((void**)&dev_i, n * sizeof(int));
			checkCUDAError("hipMalloc dev_i failed");
			hipMalloc((void**)&dev_o, n * sizeof(int));
			checkCUDAError("hipMalloc dev_o failed");
			hipMalloc((void**)&dev_e, n * sizeof(int));
			checkCUDAError("hipMalloc dev_e failed");
			hipMalloc((void**)&dev_f, n * sizeof(int));
			checkCUDAError("hipMalloc dev_f failed");
			hipMalloc((void**)&dev_t, n * sizeof(int));
			checkCUDAError("hipMalloc dev_t failed");

			int * dev_cur_i = dev_i;
			int * dev_cur_o = dev_o;
			/*
			if( (msb - lsb) % 2 == 0)
			{
				dev_cur_i = dev_i;
				dev_cur_o = dev_o;
			}
			else
			{
				dev_cur_i = dev_o;
				dev_cur_o = dev_i;
			}
			*/

			hipMemcpy(dev_cur_i,idata,n*sizeof(int),hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_cur_i failed");


			for(int i = lsb; i <= msb; i++)
			{
				radixSort(n,dev_cur_o,dev_cur_i,i);

				int * tmp = dev_cur_i;
				dev_cur_i = dev_cur_o;
				dev_cur_o = tmp;
			}


			//merge


			////////
			
			hipMemcpy(odata,dev_cur_i,n*sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_cur_o to odata failed");


			hipFree(dev_i);
			hipFree(dev_o);
			hipFree(dev_e);
			hipFree(dev_f);
			hipFree(dev_t);
		}






	}
}
